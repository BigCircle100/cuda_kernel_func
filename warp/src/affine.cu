#include "hip/hip_runtime.h"

#include "affine.h"
#include "utils.h"

// 求逆仿射变换矩阵
// getAffineTransform默认得到CV_64F，如果后续要float的话需要转换一下，因为存储方式不同不能直接按float转换
cv::Mat getAffineMatrix(const cv::Size& src_size, const cv::Size& dst_size){
  // int targetSize = 640;

  float scale_x = static_cast<float>(dst_size.width) / src_size.width;
  float scale_y = static_cast<float>(dst_size.height) / src_size.height;
  float scale = std::min(scale_x, scale_y);
  int dstHeight = static_cast<int>(src_size.height*scale);
  int dstWidth = static_cast<int>(src_size.width*scale);
  cv::Point2f dstCtr(dst_size.width/2.0f, dst_size.height/2.0f);

  std::vector<cv::Point2f> srcPoints = {
    cv::Point2f(0, 0),                    // left top
    cv::Point2f(src_size.width, 0),       // right top
    cv::Point2f(0, src_size.height)       // left bottom
  };
  std::vector<cv::Point2f> dstPoints = {
    cv::Point2f(dstCtr.x-dstWidth/2.0f, dstCtr.y-dstHeight/2.0f),
    cv::Point2f(dstCtr.x+dstWidth/2.0f, dstCtr.y-dstHeight/2.0f),
    cv::Point2f(dstCtr.x-dstWidth/2.0f, dstCtr.y+dstHeight/2.0f),
  };
  
  // cv::Mat M = cv::getAffineTransform(srcPoints, dstPoints);
  cv::Mat M = cv::getAffineTransform(dstPoints, srcPoints);
  cv::Mat floatMat;
  M.convertTo(floatMat, CV_32F);

  return floatMat;
}

// 这里是双线性插值（bilinar）
// 按照目标图片的点位p求原图坐标p'以及周围的四个点(v1~v4)，并根据v1~v4与p'的面积作为权重w，求出p的像素值
//    v1-------v2
//    | w4 | w3 |
//    |----p'---|
//    | w2 |  w1|
//    v3-------v4
// 每个thread负责同一位置的3通道
// image = Mat(height, width, channel)
__global__ 
void warp_affine_padding_kernel(
  uint8_t *src, int src_width, int src_height,
  uint8_t *dst, int dst_width, int dst_height,
  uint8_t c0, uint8_t c1, uint8_t c2, float *matrix){
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  int idy = blockDim.y*blockIdx.y + threadIdx.y;
  if (idx >= dst_width || idy >= dst_height){
    return;
  }

  // [x',y',1]^T = M*[x,y,1]^T 
  float src_x = matrix[0]*idx + matrix[1]*idy + matrix[2];
  float src_y = matrix[3]*idx + matrix[4]*idy + matrix[5];

  if (src_x >= 0 && src_x < src_width && src_y >= 0 && src_y < src_height){
    int x_low = floorf(src_x);
    int y_low = floorf(src_y);
    int x_high = (x_low + 1 < src_width) ? x_low+1 : x_low;
    int y_high = (y_low + 1 < src_height) ? y_low+1 : y_low;

    float ly = src_y - y_low;
    float lx = src_x - x_low;
    float hy = 1 - ly;
    float hx = 1 - lx;
    float w1 = hy * hx,  w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    
    uint8_t* v1 = src + y_low*src_width*3 + x_low*3;
    uint8_t* v2 = src + y_low*src_width*3 + x_high*3;
    uint8_t* v3 = src + y_high*src_width*3 + x_low*3;
    uint8_t* v4 = src + y_high*src_width*3 + x_high*3;

    c0 = w1*v1[0] + w2*v2[0] + w3*v3[0] + w4*v4[0];
    c1 = w1*v1[1] + w2*v2[1] + w3*v3[1] + w4*v4[1];
    c2 = w1*v1[2] + w2*v2[2] + w3*v3[2] + w4*v4[2];

  }
  
  uint8_t* pdst = dst + idy*dst_width*3 + idx*3;
  pdst[0] = c0;
  pdst[1] = c1;
  pdst[2] = c2;


}

cv::Mat warp_affine_padding(const cv::Mat& image, const cv::Size& size){
  cv::Mat output(size, CV_8UC3);

  uint8_t* d_src;
  uint8_t* d_dst;
  float* d_matrix;

  size_t src_size = image.cols * image.rows * 3;
  size_t dst_size = size.width * size.height * 3;
  size_t matrix_size = 6 * sizeof(float);

  auto matrix = getAffineMatrix(image.size(), size);

  checkCudaErrors(hipMalloc(&d_src, src_size));
  checkCudaErrors(hipMalloc(&d_dst, dst_size));
  checkCudaErrors(hipMalloc(&d_matrix, matrix_size));

  checkCudaErrors(hipMemcpy(d_src, image.data, src_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_matrix, matrix.ptr<float>(), matrix_size, hipMemcpyHostToDevice));

// vvvvvvvvvvv kernel part vvvvvvvvvvv

  dim3 block_size(32, 32);
  dim3 grid_size((size.width+31)/32, (size.height+31)/32);
  warp_affine_padding_kernel<<<grid_size, block_size>>>(d_src, image.cols, image.rows, 
                                                        d_dst, size.width, size.height, 114, 114, 114, d_matrix);


// ^^^^^^^^^^^ kernel part ^^^^^^^^^^^

  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(output.data, d_dst, dst_size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_src));
  checkCudaErrors(hipFree(d_dst));
  checkCudaErrors(hipFree(d_matrix));

  return output;
}
